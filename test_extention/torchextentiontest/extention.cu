#include <hip/hip_runtime.h>
#include <torch/extension.h>

std::string get_info()
{
    int cuda_version_code; hipRuntimeGetVersion(&cuda_version_code);
    std::string cuda_major_version = std::to_string(cuda_version_code / 1000);
    std::string cuda_minor_version = std::to_string((cuda_version_code % 1000) / 10);
    std::string cuda_version_info = "Cuda Version: " + std::to_string(cuda_version_code);
    // std::string cuda_version_info = "Cuda Version: " + cuda_major_version + "." + cuda_minor_version;

    std::string torch_major_version = std::to_string(TORCH_VERSION_MAJOR);
    std::string torch_minor_version = std::to_string(TORCH_VERSION_MINOR);
    std::string torch_patch_version = std::to_string(TORCH_VERSION_PATCH);
    std::string torch_version_info = "Torch Version: " + torch_major_version + "." + torch_minor_version + "." + torch_patch_version;

    return cuda_version_info + "\n" + torch_version_info;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) 
{
    m.def("get_info", &get_info);
}
